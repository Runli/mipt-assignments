#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <stdlib.h>

// includes, project
#include <cutil_inline.h>

#define MATRIX_SIZE 16
#define BLOCK_SIZE 4

__global__ void multiply_matrix(float* matrix_a, float* matrix_b, float* matrix_c) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int n = y * MATRIX_SIZE + x;

	float result = 0.0f;
	for(int i = 0; i < MATRIX_SIZE / BLOCK_SIZE; i++) {
		__shared__ float shared_a[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float shared_b[BLOCK_SIZE][BLOCK_SIZE];

		int ax = i * BLOCK_SIZE + threadIdx.x;
		int ay = blockIdx.y * BLOCK_SIZE + threadIdx.y;
		int bx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
		int by = i * BLOCK_SIZE + threadIdx.y;

		shared_a[threadIdx.y][threadIdx.x] = matrix_a[ay * MATRIX_SIZE + ax];
		shared_b[threadIdx.y][threadIdx.x] = matrix_b[by * MATRIX_SIZE + bx];

		__syncthreads();

		for(int k = 0; k < BLOCK_SIZE; k++) {
			result += shared_a[threadIdx.y][k] * shared_b[k][threadIdx.x];
		}

		__syncthreads();
	}
	matrix_c[n] = result;
}

int main() {
	size_t size = MATRIX_SIZE * MATRIX_SIZE;
	size_t byte_size = size * sizeof(float);

	float* host_matrix_a = (float*)malloc(byte_size);
	float* host_matrix_b = (float*)malloc(byte_size);
	float* host_matrix_c = (float*)malloc(byte_size);
	float* device_matrix_a;
	float* device_matrix_b;
	float* device_matrix_c;
	hipMalloc(&device_matrix_a, byte_size);
	hipMalloc(&device_matrix_b, byte_size);
	hipMalloc(&device_matrix_c, byte_size);

	FILE* input = fopen("matrix_a.txt", "r+");
	for (int i = 0; i < size; i++) {
		fscanf(input, "%f", &host_matrix_a[i]);
	}
	fclose(input);

	input = fopen("matrix_b.txt", "r+");
	for (int i = 0; i < size; i++) {
		fscanf(input, "%f", &host_matrix_b[i]);
	}
	fclose(input);

	hipMemcpy(device_matrix_a, host_matrix_a, byte_size, hipMemcpyHostToDevice);
	hipMemcpy(device_matrix_b, host_matrix_b, byte_size, hipMemcpyHostToDevice);

	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_size(MATRIX_SIZE / BLOCK_SIZE, MATRIX_SIZE / BLOCK_SIZE);
	multiply_matrix<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_matrix_c);

	hipMemcpy(host_matrix_c, device_matrix_c, byte_size, hipMemcpyDeviceToHost);

	FILE* output = fopen("matrix_c.txt", "w+");
	for (int i = 0; i < size; i++) {
		fprintf(output, "%.1f ", host_matrix_c[i]);
		if((i + 1) % MATRIX_SIZE == 0) fprintf(output, "\n");
	}
	fclose(output);

	free(host_matrix_a);
	free(host_matrix_b);
	free(host_matrix_c);
	hipFree(device_matrix_a);
	hipFree(device_matrix_b);
	hipFree(device_matrix_c);
}