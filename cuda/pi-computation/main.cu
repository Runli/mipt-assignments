#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

__global__ void compute_pi(float* result) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int n = gridDim.x * blockDim.x;

	float dx = 1.0f / (float)n;
	float x = (float)i / (float)n + dx / 2;
	float y = sqrt(1 - x * x);

	if (i < n) {
		result[i] = dx * y;
	}
}

int main() {
	int grid_size = 256;
	int block_size = 256;
	size_t size = grid_size * block_size * sizeof(float);

	float* host_memory = (float*)malloc(size);
	float* device_memory;
	hipMalloc(&device_memory, size);

	hipDeviceSynchronize();
	hipError_t status = hipGetLastError();
	if(status != hipSuccess) {
		printf("%s\n", hipGetErrorString(status));
		return 0;
	}

	compute_pi<<<grid_size, block_size>>>(device_memory);

	hipMemcpy(host_memory, device_memory, size, hipMemcpyDeviceToHost);

	float result = 0.0f;
	for (int i = 0; i < grid_size * block_size; i++) {
		result += host_memory[i];
	}

	printf("%f\n", result * 4);

	free(host_memory);
	hipFree(device_memory);
}